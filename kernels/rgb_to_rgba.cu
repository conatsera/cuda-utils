
#include <hip/hip_runtime.h>
__global__ void rgb_to_rgba_shift_segment(unsigned char* __restrict__ image_bytes, const __grid_constant__ unsigned int start_index) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x + start_index;
    const unsigned int rgb_base_addr = index * 3;
    const unsigned int rgba_base_addr = index * 4;
    
    const uchar4 rgba_pixel = make_uchar4(image_bytes[rgb_base_addr], image_bytes[rgb_base_addr+1], image_bytes[rgb_base_addr+2], 0);
    memcpy(&image_bytes[rgba_base_addr], &rgba_pixel, 4);
}
__global__ void rgb_to_rgba_shift_segment_final(unsigned char* __restrict__ image_bytes) {
    const unsigned int index = threadIdx.x;
    const unsigned int rgb_base_addr = index * 3;

    const uchar4 rgba_pixel = make_uchar4(image_bytes[rgb_base_addr], image_bytes[rgb_base_addr+1], image_bytes[rgb_base_addr+2], 0);

    __syncthreads();

    const unsigned int rgba_base_addr = index * 4;
    memcpy(&image_bytes[rgba_base_addr], &rgba_pixel, 4);
}
struct BlockMove {
    unsigned int size = 0;
    unsigned int block_size = 0;
    unsigned int position = 0;
    __device__ BlockMove(
        const unsigned int size = 0,
        const unsigned int block_size = 0,
        const unsigned int position = 0
        ) : size(size), block_size(block_size), position(position) {}
};
extern "C" __global__ void rgb_to_rgba(
        unsigned long* __restrict__ image,
        const __grid_constant__ unsigned int block_move_count,
        const __grid_constant__ unsigned int final_position,
        const BlockMove *const __restrict__ block_moves
    ) {
    unsigned char* __restrict__ image_bytes = (unsigned char*) image;

    #pragma unroll 64
    for (int i = 0; i < block_move_count; i++) {
        rgb_to_rgba_shift_segment<<<block_moves[i].size / block_moves[i].block_size, block_moves[i].block_size>>>(image_bytes, block_moves[i].position);
    }

    rgb_to_rgba_shift_segment_final<<<1, final_position>>>(image_bytes);
}

/*
printf("%u %u\n", position, free_segment_size);
while (free_segment_size > 0) {
    //printf("%u %u\n", block_size, free_segment_size);
    rgb_to_rgba_shift_segment<<<free_segment_size, 1>>>(image_bytes, position);
    free_segment_size /= 4;
    free_segment_size *= 3;
    position -= free_segment_size;
}


for (int i = 1024; i > 1; i >>= 1) {
    
    if (free_segment_size % i == 0) {
        rgb_to_rgba_shift_segment<<<free_segment_size / i, i>>>(image_bytes, position);
        free_segment_size /= 4;
        free_segment_size *= 3;
        position -= free_segment_size;
        i = 2048;
    }
}

for (int i = 1024; i > 0; i--) {
    while (free_segment_size > 0 && free_segment_size % i == 0) {
        rgb_to_rgba_shift_segment<<<free_segment_size / i, i>>>(image_bytes, position);
        free_segment_size /= 4;
        free_segment_size *= 3;
        position -= free_segment_size;
    }
}
*/
