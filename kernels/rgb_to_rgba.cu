#include "hip/hip_runtime.h"
#ifndef SIZE_PIXELS
#define SIZE_PIXELS 4096 * 3072
#endif

struct BlockMove
{
    unsigned int size = 0;
    unsigned int block_size = 0;
    unsigned int position = 0;
    __device__ constexpr BlockMove(
        unsigned int size = 0,
        unsigned int block_size = 0,
        unsigned int position = 0) : size(size), block_size(block_size), position(position) {}
};

__device__ constexpr unsigned int calculate_block_count()
{
    int free_segment_size = SIZE_PIXELS / 4;
    int i = 0;
    while (free_segment_size > 0)
    {
        i++;
        free_segment_size /= 4;
        free_segment_size *= 3;
    }
    return i;
}

constexpr const unsigned int kBlockCount = calculate_block_count();

__device__ constexpr unsigned int calculate_optimal_block_size(const unsigned int size)
{
    for (int i = 256; i > 0; i--)
    {
        if (size % i == 0)
        {
            return i;
        }
    }
}

struct BlockMoves
{
    __device__ constexpr BlockMoves() : moves()
    {
        int free_segment_size = SIZE_PIXELS / 4;
        int position = SIZE_PIXELS - free_segment_size;
        for (int i = 0; i < kBlockCount; i++)
        {
            moves[i] = BlockMove(free_segment_size, calculate_optimal_block_size(free_segment_size), position);
            free_segment_size /= 4;
            free_segment_size *= 3;
            position -= free_segment_size;
        }
    }
    BlockMove moves[kBlockCount];
};

__global__ void rgb_to_rgba_shift_segment(unsigned char* __restrict__ image_bytes, const __grid_constant__ unsigned int start_index)
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x + start_index;
    unsigned int rgb_base_addr = index * 3;
    unsigned int rgba_base_addr = index * 4;

    uchar4 rgba_pixel = make_uchar4(image_bytes[rgb_base_addr], image_bytes[rgb_base_addr + 1], image_bytes[rgb_base_addr + 2], 0);
    memcpy(&image_bytes[rgba_base_addr], &rgba_pixel, 4);
}

__global__ void rgb_to_rgba_shift_segment_final(unsigned char* __restrict__ image_bytes)
{
    unsigned int index = threadIdx.x;
    unsigned int rgb_base_addr = index * 3;

    uchar4 rgba_pixel = make_uchar4(image_bytes[rgb_base_addr], image_bytes[rgb_base_addr + 1], image_bytes[rgb_base_addr + 2], 0);

    __syncthreads();

    unsigned int rgba_base_addr = index * 4;
    memcpy(&image_bytes[rgba_base_addr], &rgba_pixel, 4);
}

extern "C" __global__ void rgb_to_rgba(unsigned char *__restrict__ image_bytes)
{
    constexpr auto block_moves = BlockMoves();

#pragma unroll
    for (int i = 0; i < kBlockCount; i++)
    {
        unsigned int block_size = block_moves.moves[i].block_size;
        //printf("%u %u %u\n", block_moves.moves[i].size, block_size, block_moves.moves[i].position);
        rgb_to_rgba_shift_segment<<<block_moves.moves[i].size / block_size, block_size>>>(image_bytes, block_moves.moves[i].position);
    }

    rgb_to_rgba_shift_segment_final<<<1, block_moves.moves[kBlockCount-1].position>>>(image_bytes);
}
